// Simplest possible XLA Custom Call Implementation

#include <hip/hip_runtime.h>

// CUDA kernel
__global__ void SimpleAddKernel(const float* a, const float* b, float* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = a[idx] + b[idx];
    }
}

// XLA Custom Call entry point
extern "C" __attribute__((visibility("default")))
void XlaGpuSimpleAdd(
    hipStream_t stream,
    void** buffers,
    const char* opaque,
    size_t opaque_len
) {
    // Extract pointers
    const float* a = (const float*)buffers[0];
    const float* b = (const float*)buffers[1];
    float* out = (float*)buffers[2];
    
    // Fixed size for testing
    const int size = 3;
    
    // Launch kernel
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;
    
    SimpleAddKernel<<<blocks, threads, 0, stream>>>(a, b, out, size);
}