#include "hip/hip_runtime.h"
// XLA Custom Call wrapper for vLLM's reshape_and_cache_flash kernel
// This file is placed in csrc/xla_ops to directly use vLLM's kernels

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <cstdint>
#include <cstring>
#include <algorithm>

// Include vLLM's Fp8 types
#ifndef USE_ROCM
  #include "../quantization/fp8/nvidia/quant_utils.cuh"
#else
  #include "../quantization/fp8/amd/quant_utils.cuh"
#endif

// Forward declare the vLLM kernel that's defined in cache_kernels.cu
namespace vllm {

template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void reshape_and_cache_flash_kernel(
    const scalar_t* __restrict__ key,
    const scalar_t* __restrict__ value,
    cache_t* __restrict__ key_cache,
    cache_t* __restrict__ value_cache,
    const int64_t* __restrict__ slot_mapping,
    const int64_t block_stride, const int64_t page_stride,
    const int64_t head_stride, const int64_t key_stride,
    const int64_t value_stride, const int num_heads, const int head_size,
    const int block_size, const float* k_scale, const float* v_scale);

}  // namespace vllm

// XLA Custom Call descriptor structure  
struct ReshapeAndCacheDescriptor {
  int64_t num_tokens;
  int64_t num_kv_heads;
  int64_t head_size;
  int64_t num_blocks;
  int64_t block_size;
  int32_t kv_cache_dtype;  // 0: auto/same as input, 1: float16, 2: bfloat16, 3: float32
  int32_t input_dtype;     // 0: float32, 1: float16, 2: bfloat16
  uint8_t has_k_scale;
  uint8_t has_v_scale;
};

extern "C" {

// XLA Custom Call implementation
void vllm_reshape_and_cache_flash_xla(
    hipStream_t stream,
    void** buffers,
    const char* opaque,
    size_t opaque_len) {
  
  // Parse descriptor
  ReshapeAndCacheDescriptor descriptor;
  memcpy(&descriptor, opaque, sizeof(ReshapeAndCacheDescriptor));
  
  // Extract buffers in the expected order
  void* key_cache_buffer = buffers[0];
  void* value_cache_buffer = buffers[1];
  const void* key_buffer = buffers[2];
  const void* value_buffer = buffers[3];
  const void* slot_mapping_buffer = buffers[4];
  
  // Handle optional scale buffers correctly
  int buffer_idx = 5;
  const float* k_scale_buffer = nullptr;
  const float* v_scale_buffer = nullptr;
  
  if (descriptor.has_k_scale) {
    k_scale_buffer = static_cast<const float*>(buffers[buffer_idx++]);
  }
  if (descriptor.has_v_scale) {
    v_scale_buffer = static_cast<const float*>(buffers[buffer_idx]);
  }
  
  // Calculate strides (matching vLLM's layout for reshape_and_cache_flash)
  int64_t key_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t value_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t block_stride = descriptor.block_size * descriptor.num_kv_heads * descriptor.head_size;
  int64_t page_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t head_stride = descriptor.head_size;
  
  // Launch kernel with 1D grid and 1D block (exactly like vLLM)
  dim3 grid(descriptor.num_tokens);
  dim3 block(std::min((int64_t)512, descriptor.num_kv_heads * descriptor.head_size));
  
  // Dispatch based on input and cache data types
  // Call vLLM's kernel directly
  if (descriptor.input_dtype == 0) {  // float32 input
    if (descriptor.kv_cache_dtype == 0 || descriptor.kv_cache_dtype == 3) {
      // float32 -> float32
      vllm::reshape_and_cache_flash_kernel<float, float, vllm::Fp8KVCacheDataType::kAuto>
          <<<grid, block, 0, stream>>>(
          static_cast<const float*>(key_buffer),
          static_cast<const float*>(value_buffer),
          static_cast<float*>(key_cache_buffer),
          static_cast<float*>(value_cache_buffer),
          static_cast<const int64_t*>(slot_mapping_buffer),
          block_stride, page_stride, head_stride,
          key_stride, value_stride,
          descriptor.num_kv_heads, descriptor.head_size, descriptor.block_size,
          k_scale_buffer, v_scale_buffer);
    } else if (descriptor.kv_cache_dtype == 1) {
      // float32 -> float16
      vllm::reshape_and_cache_flash_kernel<float, __half, vllm::Fp8KVCacheDataType::kAuto>
          <<<grid, block, 0, stream>>>(
          static_cast<const float*>(key_buffer),
          static_cast<const float*>(value_buffer),
          static_cast<__half*>(key_cache_buffer),
          static_cast<__half*>(value_cache_buffer),
          static_cast<const int64_t*>(slot_mapping_buffer),
          block_stride, page_stride, head_stride,
          key_stride, value_stride,
          descriptor.num_kv_heads, descriptor.head_size, descriptor.block_size,
          k_scale_buffer, v_scale_buffer);
    } else if (descriptor.kv_cache_dtype == 2) {
      // float32 -> bfloat16
      vllm::reshape_and_cache_flash_kernel<float, __hip_bfloat16, vllm::Fp8KVCacheDataType::kAuto>
          <<<grid, block, 0, stream>>>(
          static_cast<const float*>(key_buffer),
          static_cast<const float*>(value_buffer),
          static_cast<__hip_bfloat16*>(key_cache_buffer),
          static_cast<__hip_bfloat16*>(value_cache_buffer),
          static_cast<const int64_t*>(slot_mapping_buffer),
          block_stride, page_stride, head_stride,
          key_stride, value_stride,
          descriptor.num_kv_heads, descriptor.head_size, descriptor.block_size,
          k_scale_buffer, v_scale_buffer);
    }
  } else if (descriptor.input_dtype == 1) {  // float16 input
    if (descriptor.kv_cache_dtype == 0 || descriptor.kv_cache_dtype == 1) {
      // float16 -> float16
      vllm::reshape_and_cache_flash_kernel<__half, __half, vllm::Fp8KVCacheDataType::kAuto>
          <<<grid, block, 0, stream>>>(
          static_cast<const __half*>(key_buffer),
          static_cast<const __half*>(value_buffer),
          static_cast<__half*>(key_cache_buffer),
          static_cast<__half*>(value_cache_buffer),
          static_cast<const int64_t*>(slot_mapping_buffer),
          block_stride, page_stride, head_stride,
          key_stride, value_stride,
          descriptor.num_kv_heads, descriptor.head_size, descriptor.block_size,
          nullptr, nullptr);  // No scaling for fp16
    }
  } else if (descriptor.input_dtype == 2) {  // bfloat16 input
    if (descriptor.kv_cache_dtype == 0 || descriptor.kv_cache_dtype == 2) {
      // bfloat16 -> bfloat16
      vllm::reshape_and_cache_flash_kernel<__hip_bfloat16, __hip_bfloat16, vllm::Fp8KVCacheDataType::kAuto>
          <<<grid, block, 0, stream>>>(
          static_cast<const __hip_bfloat16*>(key_buffer),
          static_cast<const __hip_bfloat16*>(value_buffer),
          static_cast<__hip_bfloat16*>(key_cache_buffer),
          static_cast<__hip_bfloat16*>(value_cache_buffer),
          static_cast<const int64_t*>(slot_mapping_buffer),
          block_stride, page_stride, head_stride,
          key_stride, value_stride,
          descriptor.num_kv_heads, descriptor.head_size, descriptor.block_size,
          nullptr, nullptr);  // No scaling for bf16
    }
  }
}

} // extern "C"