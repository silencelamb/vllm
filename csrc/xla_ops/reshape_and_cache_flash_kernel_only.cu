// Standalone vLLM reshape_and_cache_flash kernel for XLA
// This version includes only the kernel, not the full cache_kernels.cu

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <cstdint>
#include <cstring>
#include <algorithm>
#include <stdio.h>

namespace vllm {

// Simplified Fp8KVCacheDataType enum
enum Fp8KVCacheDataType {
  kAuto = 0,
  kFp8E4M3 = 1,
  kFp8E5M2 = 2,
};

// The reshape_and_cache_flash kernel from vLLM
template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void reshape_and_cache_flash_kernel(
    const scalar_t* __restrict__ key,    // [num_tokens, num_heads, head_size]
    const scalar_t* __restrict__ value,  // [num_tokens, num_heads, head_size]
    cache_t* __restrict__ key_cache,     // [num_blocks, block_size, num_heads,
                                         // head_size]
    cache_t* __restrict__ value_cache,   // [num_blocks, block_size, num_heads,
                                         // head_size]
    const int64_t* __restrict__ slot_mapping,  // [num_tokens]
    const int64_t block_stride, const int64_t page_stride,
    const int64_t head_stride, const int64_t key_stride,
    const int64_t value_stride, const int num_heads, const int head_size,
    const int block_size, const float* k_scale, const float* v_scale) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];

  // if (blockIdx.x == 0 && threadIdx.x == 0) {
  //   for (int64_t i = 0; i < 4; ++i) {
  //     printf("slot[%ld] = %ld\n", i, slot_mapping[i]);
  //   }
  // }
  // NOTE: slot_idx can be -1 if the token is padded
  if (slot_idx < 0) {
    return;
  }
  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;
  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int64_t src_key_idx = token_idx * key_stride + i;
    const int64_t src_value_idx = token_idx * value_stride + i;
    const int head_idx = i / head_size;
    const int head_offset = i % head_size;
    const int64_t tgt_key_value_idx = block_idx * block_stride +
                                      block_offset * page_stride +
                                      head_idx * head_stride + head_offset;
    scalar_t tgt_key = key[src_key_idx];
    scalar_t tgt_value = value[src_value_idx];
    // printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);
    // printf(
    //     "block_idx: %ld, block_offset: %ld, head_idx: %d, "
    //     "head_offset: %d, tgt_key_value_idx: %ld, tgt_key: %f, tgt_value:
    //     %f\n", block_idx, block_offset, head_idx, head_offset,
    //     tgt_key_value_idx, static_cast<float>(tgt_key),
    //     static_cast<float>(tgt_value));

    // For kAuto mode, no fp8 conversion
    if (kv_dt == Fp8KVCacheDataType::kAuto) {
      // Apply scaling if provided
      if (k_scale != nullptr || v_scale != nullptr) {
        // Convert to float for scaling, then back to target type
        float key_float = static_cast<float>(tgt_key);
        float value_float = static_cast<float>(tgt_value);

        if (k_scale != nullptr) {
          key_float = key_float * k_scale[0];
        }
        if (v_scale != nullptr) {
          value_float = value_float * v_scale[0];
        }

        key_cache[tgt_key_value_idx] = cache_t(key_float);
        value_cache[tgt_key_value_idx] = cache_t(value_float);
      } else {
        // No scaling, direct copy
        // printf("No scaling applied, copying directly.\n");
        key_cache[tgt_key_value_idx] = cache_t(tgt_key);
        value_cache[tgt_key_value_idx] = cache_t(tgt_value);
      }
    }
  }

  __syncthreads();
}

}  // namespace vllm

// XLA Custom Call descriptor structure
struct ReshapeAndCacheDescriptor {
  int64_t num_tokens;
  int64_t num_kv_heads;
  int64_t head_size;
  int64_t num_blocks;
  int64_t block_size;
  int32_t kv_cache_dtype;  // 0: auto/same as input, 1: float16, 2: bfloat16, 3:
                           // float32
  int32_t input_dtype;     // 0: float32, 1: float16, 2: bfloat16
  uint8_t has_k_scale;
  uint8_t has_v_scale;
};

extern "C" {

// XLA Custom Call implementation
void vllm_reshape_and_cache_flash_xla(hipStream_t stream, void** buffers,
                                      const char* opaque, size_t opaque_len) {
  // Parse descriptor
  ReshapeAndCacheDescriptor descriptor;
  memcpy(&descriptor, opaque, sizeof(ReshapeAndCacheDescriptor));

  // Extract buffers in the expected order
  void* key_cache_buffer = buffers[0];
  void* value_cache_buffer = buffers[1];
  const void* key_buffer = buffers[2];
  const void* value_buffer = buffers[3];
  const void* slot_mapping_buffer = buffers[4];

  // Handle optional scale buffers correctly
  int buffer_idx = 5;
  const float* k_scale_buffer = nullptr;
  const float* v_scale_buffer = nullptr;

  if (descriptor.has_k_scale) {
    k_scale_buffer = static_cast<const float*>(buffers[buffer_idx++]);
  }
  if (descriptor.has_v_scale) {
    v_scale_buffer = static_cast<const float*>(buffers[buffer_idx]);
  }

  // Calculate strides (matching vLLM's layout for reshape_and_cache_flash)
  int64_t key_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t value_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t block_stride =
      descriptor.block_size * descriptor.num_kv_heads * descriptor.head_size;
  int64_t page_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t head_stride = descriptor.head_size;

  // Launch kernel with 1D grid and 1D block (exactly like vLLM)
  dim3 grid(descriptor.num_tokens);
  dim3 block(
      std::min((int64_t)512, descriptor.num_kv_heads * descriptor.head_size));

  // Dispatch based on input and cache data types
  if (descriptor.input_dtype == 0) {  // float32 input
    if (descriptor.kv_cache_dtype == 0 || descriptor.kv_cache_dtype == 3) {
      // float32 -> float32
      vllm::reshape_and_cache_flash_kernel<float, float,
                                           vllm::Fp8KVCacheDataType::kAuto>
          <<<grid, block, 0, stream>>>(
              static_cast<const float*>(key_buffer),
              static_cast<const float*>(value_buffer),
              static_cast<float*>(key_cache_buffer),
              static_cast<float*>(value_cache_buffer),
              static_cast<const int64_t*>(slot_mapping_buffer), block_stride,
              page_stride, head_stride, key_stride, value_stride,
              descriptor.num_kv_heads, descriptor.head_size,
              descriptor.block_size, k_scale_buffer, v_scale_buffer);
    } else if (descriptor.kv_cache_dtype == 1) {
      // float32 -> float16
      vllm::reshape_and_cache_flash_kernel<float, __half,
                                           vllm::Fp8KVCacheDataType::kAuto>
          <<<grid, block, 0, stream>>>(
              static_cast<const float*>(key_buffer),
              static_cast<const float*>(value_buffer),
              static_cast<__half*>(key_cache_buffer),
              static_cast<__half*>(value_cache_buffer),
              static_cast<const int64_t*>(slot_mapping_buffer), block_stride,
              page_stride, head_stride, key_stride, value_stride,
              descriptor.num_kv_heads, descriptor.head_size,
              descriptor.block_size, k_scale_buffer, v_scale_buffer);
    } else if (descriptor.kv_cache_dtype == 2) {
      // float32 -> bfloat16
      vllm::reshape_and_cache_flash_kernel<float, __hip_bfloat16,
                                           vllm::Fp8KVCacheDataType::kAuto>
          <<<grid, block, 0, stream>>>(
              static_cast<const float*>(key_buffer),
              static_cast<const float*>(value_buffer),
              static_cast<__hip_bfloat16*>(key_cache_buffer),
              static_cast<__hip_bfloat16*>(value_cache_buffer),
              static_cast<const int64_t*>(slot_mapping_buffer), block_stride,
              page_stride, head_stride, key_stride, value_stride,
              descriptor.num_kv_heads, descriptor.head_size,
              descriptor.block_size, k_scale_buffer, v_scale_buffer);
    }
  } else if (descriptor.input_dtype == 1) {  // float16 input
    if (descriptor.kv_cache_dtype == 0 || descriptor.kv_cache_dtype == 1) {
      // float16 -> float16
      vllm::reshape_and_cache_flash_kernel<__half, __half,
                                           vllm::Fp8KVCacheDataType::kAuto>
          <<<grid, block, 0, stream>>>(
              static_cast<const __half*>(key_buffer),
              static_cast<const __half*>(value_buffer),
              static_cast<__half*>(key_cache_buffer),
              static_cast<__half*>(value_cache_buffer),
              static_cast<const int64_t*>(slot_mapping_buffer), block_stride,
              page_stride, head_stride, key_stride, value_stride,
              descriptor.num_kv_heads, descriptor.head_size,
              descriptor.block_size, nullptr, nullptr);  // No scaling for fp16
    }
  } else if (descriptor.input_dtype == 2) {  // bfloat16 input
    if (descriptor.kv_cache_dtype == 0 || descriptor.kv_cache_dtype == 2) {
      // bfloat16 -> bfloat16
      vllm::reshape_and_cache_flash_kernel<__hip_bfloat16, __hip_bfloat16,
                                           vllm::Fp8KVCacheDataType::kAuto>
          <<<grid, block, 0, stream>>>(
              static_cast<const __hip_bfloat16*>(key_buffer),
              static_cast<const __hip_bfloat16*>(value_buffer),
              static_cast<__hip_bfloat16*>(key_cache_buffer),
              static_cast<__hip_bfloat16*>(value_cache_buffer),
              static_cast<const int64_t*>(slot_mapping_buffer), block_stride,
              page_stride, head_stride, key_stride, value_stride,
              descriptor.num_kv_heads, descriptor.head_size,
              descriptor.block_size, nullptr, nullptr);  // No scaling for bf16
    }
  }
}

}  // extern "C"