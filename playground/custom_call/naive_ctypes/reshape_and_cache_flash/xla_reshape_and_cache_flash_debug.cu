// Debug version of reshape_and_cache_flash for XLA Custom Call
// Includes printf debugging to understand what's happening

#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstring>
#include <algorithm>
#include <cstdio>

// XLA Custom Call descriptor structure  
struct ReshapeAndCacheDescriptor {
  int64_t num_tokens;
  int64_t num_kv_heads;
  int64_t head_size;
  int64_t num_blocks;
  int64_t block_size;
  int32_t kv_cache_dtype;  // 0: auto/float32, 1: float16, 2: bfloat16
  uint8_t has_k_scale;
  uint8_t has_v_scale;
};

extern "C" {

// Debug kernel with printf
__global__ void reshape_and_cache_kernel_debug(
    float* key_cache,
    float* value_cache,
    const float* key,
    const float* value,
    const int64_t* slot_mapping,
    const float* k_scale,
    const float* v_scale,
    int num_tokens,
    int num_kv_heads,
    int head_size,
    int block_size) {
  
  int token_idx = blockIdx.x;
  int head_idx = blockIdx.y;
  int dim_idx = threadIdx.x;
  
  // Debug print from first thread
  if (token_idx == 0 && head_idx == 0 && dim_idx == 0) {
    printf("Kernel launched with: tokens=%d, heads=%d, head_size=%d, block_size=%d\n", 
           num_tokens, num_kv_heads, head_size, block_size);
    printf("k_scale=%p, v_scale=%p\n", k_scale, v_scale);
    if (k_scale) printf("k_scale[0]=%f\n", k_scale[0]);
    if (v_scale) printf("v_scale[0]=%f\n", v_scale[0]);
  }
  
  if (token_idx >= num_tokens || head_idx >= num_kv_heads || dim_idx >= head_size) {
    return;
  }
  
  // Get slot index for this token
  int64_t slot_idx = slot_mapping[token_idx];
  
  // Debug print slot mapping
  if (head_idx == 0 && dim_idx == 0) {
    printf("Token %d -> slot %lld\n", token_idx, (long long)slot_idx);
  }
  
  if (slot_idx < 0) {
    return;  // Invalid slot
  }
  
  // Calculate block and position within block
  int block_idx = slot_idx / block_size;
  int block_offset = slot_idx % block_size;
  
  // Calculate input indices
  int input_idx = token_idx * num_kv_heads * head_size + head_idx * head_size + dim_idx;
  
  // Calculate cache indices - layout: [num_blocks, block_size, num_kv_heads, head_size]
  int cache_idx = block_idx * (block_size * num_kv_heads * head_size) +
                  block_offset * (num_kv_heads * head_size) +
                  head_idx * head_size +
                  dim_idx;
  
  // Get key and value
  float key_val = key[input_idx];
  float value_val = value[input_idx];
  
  // Debug print before scaling
  if (token_idx == 0 && head_idx == 0 && dim_idx == 0) {
    printf("Before scaling: key_val=%f, value_val=%f\n", key_val, value_val);
  }
  
  // Apply scaling if provided
  if (k_scale != nullptr) {
    key_val *= k_scale[0];
  }
  if (v_scale != nullptr) {
    value_val *= v_scale[0];
  }
  
  // Debug print after scaling
  if (token_idx == 0 && head_idx == 0 && dim_idx == 0) {
    printf("After scaling: key_val=%f, value_val=%f\n", key_val, value_val);
    printf("Writing to cache_idx=%d\n", cache_idx);
  }
  
  // Write to cache
  key_cache[cache_idx] = key_val;
  value_cache[cache_idx] = value_val;
}

// XLA Custom Call implementation
void reshape_and_cache_flash_xla_custom_call(
    hipStream_t stream,
    void** buffers,
    const char* opaque,
    size_t opaque_len) {
  
  // Parse descriptor
  ReshapeAndCacheDescriptor descriptor;
  memcpy(&descriptor, opaque, sizeof(ReshapeAndCacheDescriptor));
  
  printf("Custom call invoked with %lld tokens, %lld heads, %lld head_size\n",
         (long long)descriptor.num_tokens, 
         (long long)descriptor.num_kv_heads,
         (long long)descriptor.head_size);
  printf("has_k_scale=%d, has_v_scale=%d\n", descriptor.has_k_scale, descriptor.has_v_scale);
  
  // Extract buffers in the expected order
  // For XLA custom calls, the output buffers come first, then inputs
  void* key_cache_buffer = buffers[0];
  void* value_cache_buffer = buffers[1];
  const void* key_buffer = buffers[2];
  const void* value_buffer = buffers[3];
  const void* slot_mapping_buffer = buffers[4];
  
  // Scale buffers come after fixed inputs
  int next_buffer = 5;
  const void* k_scale_buffer = nullptr;
  const void* v_scale_buffer = nullptr;
  
  if (descriptor.has_k_scale) {
    k_scale_buffer = buffers[next_buffer++];
  }
  if (descriptor.has_v_scale) {
    v_scale_buffer = buffers[next_buffer++];
  }
  
  printf("Buffer pointers: key_cache=%p, value_cache=%p, key=%p, value=%p\n",
         key_cache_buffer, value_cache_buffer, key_buffer, value_buffer);
  printf("slot_mapping=%p, k_scale=%p, v_scale=%p\n",
         slot_mapping_buffer, k_scale_buffer, v_scale_buffer);
  
  // For now, only handle float32
  if (descriptor.kv_cache_dtype == 0) {  // float32/auto
    // Launch kernel with 3D grid
    dim3 grid(descriptor.num_tokens, descriptor.num_kv_heads, 1);
    dim3 block(std::min((int64_t)256, descriptor.head_size), 1, 1);
    
    printf("Launching kernel with grid(%d,%d,1) block(%d,1,1)\n",
           (int)descriptor.num_tokens, (int)descriptor.num_kv_heads,
           (int)std::min((int64_t)256, descriptor.head_size));
    
    reshape_and_cache_kernel_debug<<<grid, block, 0, stream>>>(
        static_cast<float*>(key_cache_buffer),
        static_cast<float*>(value_cache_buffer),
        static_cast<const float*>(key_buffer),
        static_cast<const float*>(value_buffer),
        static_cast<const int64_t*>(slot_mapping_buffer),
        static_cast<const float*>(k_scale_buffer),
        static_cast<const float*>(v_scale_buffer),
        descriptor.num_tokens,
        descriptor.num_kv_heads,
        descriptor.head_size,
        descriptor.block_size
    );
    
    // Sync for debugging
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      printf("CUDA error: %s\n", hipGetErrorString(err));
    }
  }
}

} // extern "C"