// Debug version to understand buffer layout
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstring>
#include <cstdio>
#include <algorithm>

struct ReshapeAndCacheDescriptor {
  int64_t num_tokens;
  int64_t num_kv_heads;
  int64_t head_size;
  int64_t num_blocks;
  int64_t block_size;
  int32_t kv_cache_dtype;
  uint8_t has_k_scale;
  uint8_t has_v_scale;
};

extern "C" {

__global__ void debug_kernel(float* out, const float* in, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size && idx < 4) {
    // Just copy first 4 elements to verify we're getting the right data
    out[idx] = in[idx];
  }
}

void reshape_and_cache_flash_xla_custom_call(
    hipStream_t stream,
    void** buffers,
    const char* opaque,
    size_t opaque_len) {
  
  // Parse descriptor
  ReshapeAndCacheDescriptor descriptor;
  memcpy(&descriptor, opaque, sizeof(ReshapeAndCacheDescriptor));
  
  printf("DEBUG: num_tokens=%ld, num_kv_heads=%ld, head_size=%ld\n", 
         descriptor.num_tokens, descriptor.num_kv_heads, descriptor.head_size);
  printf("DEBUG: num_blocks=%ld, block_size=%ld\n",
         descriptor.num_blocks, descriptor.block_size);
  
  // Test different buffer arrangements
  // Let's try to understand what XLA is passing us
  
  // Try 1: Assume inputs first, outputs last
  const float* key = (const float*)buffers[0];
  const float* value = (const float*)buffers[1];
  float* key_cache_out = nullptr;
  float* value_cache_out = nullptr;
  
  // Count total buffers (5 inputs + optional scales + 2 outputs)
  int total_inputs = 5;  // key, value, key_cache, value_cache, slot_mapping
  if (descriptor.has_k_scale) total_inputs++;
  if (descriptor.has_v_scale) total_inputs++;
  
  // Outputs should be at the end
  key_cache_out = (float*)buffers[total_inputs];
  value_cache_out = (float*)buffers[total_inputs + 1];
  
  printf("DEBUG: Total inputs=%d, key_cache_out at index %d\n", total_inputs, total_inputs);
  
  // Simple test: just copy first elements from key to key_cache
  int test_size = descriptor.num_kv_heads * descriptor.head_size;
  dim3 block(256);
  dim3 grid((test_size + 255) / 256);
  
  debug_kernel<<<grid, block, 0, stream>>>(key_cache_out, key, test_size);
  debug_kernel<<<grid, block, 0, stream>>>(value_cache_out, value, test_size);
  
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
  }
}

} // extern "C"