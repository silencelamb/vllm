// XLA Custom Call wrapper that directly uses vLLM's reshape_and_cache_flash
// kernel

#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstring>
#include <algorithm>
#include <stdio.h>

// Forward declare the vLLM kernel that we'll link against
namespace vllm {

enum Fp8KVCacheDataType {
  kAuto = 0,
  kFp8E4M3 = 1,
  kFp8E5M2 = 2,
};

// The actual kernel is defined in csrc/cache_kernels.cu
template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void reshape_and_cache_flash_kernel(
    const scalar_t* __restrict__ key, const scalar_t* __restrict__ value,
    cache_t* __restrict__ key_cache, cache_t* __restrict__ value_cache,
    const int64_t* __restrict__ slot_mapping, const int64_t block_stride,
    const int64_t page_stride, const int64_t head_stride,
    const int64_t key_stride, const int64_t value_stride, const int num_heads,
    const int head_size, const int block_size, const float* k_scale,
    const float* v_scale);

}  // namespace vllm

// XLA Custom Call descriptor structure
struct ReshapeAndCacheDescriptor {
  int64_t num_tokens;
  int64_t num_kv_heads;
  int64_t head_size;
  int64_t num_blocks;
  int64_t block_size;
  int32_t kv_cache_dtype;  // 0: auto/same as input
  uint8_t has_k_scale;
  uint8_t has_v_scale;
};

extern "C" {

// XLA Custom Call implementation
void reshape_and_cache_flash_xla_custom_call(hipStream_t stream,
                                             void** buffers, const char* opaque,
                                             size_t opaque_len) {
  // Parse descriptor
  ReshapeAndCacheDescriptor descriptor;
  memcpy(&descriptor, opaque, sizeof(ReshapeAndCacheDescriptor));
  printf("ReshapeAndCacheDescriptor:\n");
  printf("  num_tokens: %ld\n", descriptor.num_tokens);
  printf("  num_kv_heads: %ld\n", descriptor.num_kv_heads);
  printf("  head_size: %ld\n", descriptor.head_size);
  printf("  num_blocks: %ld\n", descriptor.num_blocks);
  printf("  block_size: %ld\n", descriptor.block_size);
  printf("  kv_cache_dtype: %d\n", descriptor.kv_cache_dtype);
  printf("  has_k_scale: %d\n", descriptor.has_k_scale);
  printf("  has_v_scale: %d\n", descriptor.has_v_scale);

  // Extract buffers in the expected order
  // IMPORTANT: On GPU, the LAST num_outputs buffers are outputs!
  // With num_outputs=2, buffer order is:
  // Inputs:
  // - buffers[0] = input key
  // - buffers[1] = input value
  // - buffers[2] = input key_cache (for reading)
  // - buffers[3] = input value_cache (for reading)
  // - buffers[4] = input slot_mapping
  // - buffers[5] = input k_scale (if has_k_scale)
  // - buffers[6] = input v_scale (if has_v_scale)
  // Outputs (last 2):
  // - buffers[7 or N-2] = output key_cache (to write, same tensor as input for
  // in-place)
  // - buffers[8 or N-1] = output value_cache (to write, same tensor as input
  // for in-place)

  const void* key_buffer = buffers[0];           // input key
  const void* value_buffer = buffers[1];         // input value
  const void* slot_mapping_buffer = buffers[4];  // input slot_mapping

  // Calculate total number of buffers
  int num_input_buffers =
      5;  // key, value, key_cache, value_cache, slot_mapping
  // Handle optional scale buffers correctly
  const float* k_scale_buffer = nullptr;
  const float* v_scale_buffer = nullptr;

  int scale_buffer_idx = num_input_buffers;
  if (descriptor.has_k_scale) {
    k_scale_buffer = static_cast<const float*>(buffers[scale_buffer_idx++]);
  }
  if (descriptor.has_v_scale) {
    v_scale_buffer = static_cast<const float*>(buffers[scale_buffer_idx++]);
  }

  // Output buffers are at the end, after all input buffers including scales
  int output_buffer_start = scale_buffer_idx;
  void* key_cache_buffer = buffers[output_buffer_start];      // output key_cache
  void* value_cache_buffer = buffers[output_buffer_start + 1];  // output value_cache

  for (int i = 0; i < num_input_buffers; ++i) {
    printf("buffers[%d]: %p\n", i, buffers[i]);
  }

  printf("key_cache_buffer: %p, value_cache_buffer: %p\n", key_cache_buffer,
         value_cache_buffer);

  printf("buffers[2]: %p, buffers[3]: %p\n", buffers[2], buffers[3]);

  // Calculate strides (matching vLLM's layout for reshape_and_cache_flash)
  int64_t key_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t value_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t block_stride =
      descriptor.block_size * descriptor.num_kv_heads * descriptor.head_size;
  int64_t page_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t head_stride = descriptor.head_size;

  // Launch kernel with 1D grid and 1D block (exactly like vLLM)
  dim3 grid(descriptor.num_tokens);
  dim3 block(
      std::min((int64_t)512, descriptor.num_kv_heads * descriptor.head_size));

  // For now, only handle float32 case
  // Call vLLM's kernel directly - it will be linked from the compiled vLLM
  // library
  vllm::reshape_and_cache_flash_kernel<float, float,
                                       vllm::Fp8KVCacheDataType::kAuto>
      <<<grid, block, 0, stream>>>(
          static_cast<const float*>(key_buffer),
          static_cast<const float*>(value_buffer),
          static_cast<float*>(key_cache_buffer),
          static_cast<float*>(value_cache_buffer),
          static_cast<const int64_t*>(slot_mapping_buffer), block_stride,
          page_stride, head_stride, key_stride, value_stride,
          descriptor.num_kv_heads, descriptor.head_size, descriptor.block_size,
          k_scale_buffer, v_scale_buffer);
  
  // Ensure kernel completes before returning
  hipError_t err = hipStreamSynchronize(stream);
  if (err != hipSuccess) {
    printf("CUDA error in reshape_and_cache_flash: %s\n", hipGetErrorString(err));
  }
}

}  // extern "C"