// XLA Custom Call wrapper that directly uses vLLM's reshape_and_cache_flash kernel

#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstring>
#include <algorithm>

// Forward declare the vLLM kernel that we'll link against
namespace vllm {

enum Fp8KVCacheDataType {
  kAuto = 0,
  kFp8E4M3 = 1,
  kFp8E5M2 = 2,
};

// The actual kernel is defined in csrc/cache_kernels.cu
template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void reshape_and_cache_flash_kernel(
    const scalar_t* __restrict__ key,
    const scalar_t* __restrict__ value,
    cache_t* __restrict__ key_cache,
    cache_t* __restrict__ value_cache,
    const int64_t* __restrict__ slot_mapping,
    const int64_t block_stride, const int64_t page_stride,
    const int64_t head_stride, const int64_t key_stride,
    const int64_t value_stride, const int num_heads, const int head_size,
    const int block_size, const float* k_scale, const float* v_scale);

}  // namespace vllm

// XLA Custom Call descriptor structure  
struct ReshapeAndCacheDescriptor {
  int64_t num_tokens;
  int64_t num_kv_heads;
  int64_t head_size;
  int64_t num_blocks;
  int64_t block_size;
  int32_t kv_cache_dtype;  // 0: auto/same as input
  uint8_t has_k_scale;
  uint8_t has_v_scale;
};

extern "C" {

// XLA Custom Call implementation
void reshape_and_cache_flash_xla_custom_call(
    hipStream_t stream,
    void** buffers,
    const char* opaque,
    size_t opaque_len) {
  
  // Parse descriptor
  ReshapeAndCacheDescriptor descriptor;
  memcpy(&descriptor, opaque, sizeof(ReshapeAndCacheDescriptor));
  
  // Extract buffers in the expected order
  // IMPORTANT: On GPU, the LAST num_outputs buffers are outputs!
  // With num_outputs=2, buffer order is:
  // Inputs:
  // - buffers[0] = input key
  // - buffers[1] = input value
  // - buffers[2] = input key_cache
  // - buffers[3] = input value_cache
  // - buffers[4] = input slot_mapping
  // - buffers[5] = input k_scale (if has_k_scale)
  // - buffers[6] = input v_scale (if has_v_scale)
  // Outputs (last 2):
  // - buffers[N-2] = output key_cache (same tensor as input for in-place)
  // - buffers[N-1] = output value_cache (same tensor as input for in-place)
  
  const void* key_buffer = buffers[0];      // input key
  const void* value_buffer = buffers[1];    // input value
  void* key_cache_buffer = buffers[2];      // input key_cache (modified in-place)
  void* value_cache_buffer = buffers[3];    // input value_cache (modified in-place)
  const void* slot_mapping_buffer = buffers[4];  // input slot_mapping
  
  // Handle optional scale buffers correctly
  int buffer_idx = 5;  // Start after slot_mapping
  const float* k_scale_buffer = nullptr;
  const float* v_scale_buffer = nullptr;
  
  if (descriptor.has_k_scale) {
    k_scale_buffer = static_cast<const float*>(buffers[buffer_idx++]);
  }
  if (descriptor.has_v_scale) {
    v_scale_buffer = static_cast<const float*>(buffers[buffer_idx]);
  }
  
  // Calculate strides (matching vLLM's layout for reshape_and_cache_flash)
  int64_t key_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t value_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t block_stride = descriptor.block_size * descriptor.num_kv_heads * descriptor.head_size;
  int64_t page_stride = descriptor.num_kv_heads * descriptor.head_size;
  int64_t head_stride = descriptor.head_size;
  
  // Launch kernel with 1D grid and 1D block (exactly like vLLM)
  dim3 grid(descriptor.num_tokens);
  dim3 block(std::min((int64_t)512, descriptor.num_kv_heads * descriptor.head_size));
  
  // For now, only handle float32 case
  // Call vLLM's kernel directly - it will be linked from the compiled vLLM library
  vllm::reshape_and_cache_flash_kernel<float, float, vllm::Fp8KVCacheDataType::kAuto>
      <<<grid, block, 0, stream>>>(
      static_cast<const float*>(key_buffer),
      static_cast<const float*>(value_buffer),
      static_cast<float*>(key_cache_buffer),
      static_cast<float*>(value_cache_buffer),
      static_cast<const int64_t*>(slot_mapping_buffer),
      block_stride, page_stride, head_stride,
      key_stride, value_stride,
      descriptor.num_kv_heads, descriptor.head_size, descriptor.block_size,
      k_scale_buffer, v_scale_buffer);
}

} // extern "C"