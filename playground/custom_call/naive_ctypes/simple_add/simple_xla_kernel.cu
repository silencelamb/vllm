// Simplest possible XLA Custom Call Implementation

#include <hip/hip_runtime.h>
#include <cstddef>  // for size_t
#include <cstdint>  // for int32_t

// CUDA kernel
__global__ void SimpleAddKernel(const float* a, const float* b, float* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = a[idx] + b[idx];
    }
}

// XLA Custom Call entry point
extern "C" __attribute__((visibility("default")))
void XlaGpuSimpleAdd(
    hipStream_t stream,
    void** buffers,
    const char* opaque,
    size_t opaque_len
) {
    // Extract pointers
    const float* a = (const float*)buffers[0];
    const float* b = (const float*)buffers[1];
    float* out = (float*)buffers[2];
    
    // Extract size from opaque data
    // opaque contains the element count as int32
    const int size = opaque_len >= sizeof(int) ? *(const int*)opaque : 0;
    if (size == 0) return;  // Safety check
    
    // Launch kernel
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;
    
    SimpleAddKernel<<<blocks, threads, 0, stream>>>(a, b, out, size);
}