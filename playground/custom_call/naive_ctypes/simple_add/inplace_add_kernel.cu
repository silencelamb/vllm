// In-place add kernel for testing buffer donor optimization
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>

extern "C" {

// In-place add: output += input
// This modifies output in-place instead of creating new memory
__global__ void inplace_add_kernel(const float* input, float* output,
                                   int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    output[idx] += input[idx];
  }
}

// XLA Custom Call that does in-place addition
void XlaGpuInplaceAdd(hipStream_t stream, void** buffers, const char* opaque,
                      size_t opaque_len) {
  // Parse size from opaque data (int32 in little-endian)
  int32_t size;
  memcpy(&size, opaque, sizeof(int32_t));

  // XLA GPU buffer ordering:
  // - Inputs come first: buffers[0] = input
  // - Output comes last: buffers[1] = output (for in-place, same as input)
  const float* input = static_cast<const float*>(buffers[0]);
  float* output = static_cast<float*>(buffers[1]);

  // Print buffer addresses to verify in-place optimization
  printf("InplaceAdd - Input buffer:  %p\n", input);
  printf("InplaceAdd - Output buffer: %p\n", output);
  printf("buffer[2]: %p\n", buffers[2]);
  printf("InplaceAdd - Same buffer (in-place): %s\n",
         (output == buffers[2]) ? "YES" : "NO");

  // Launch kernel
  int threads = 256;
  int blocks = (size + threads - 1) / threads;
  inplace_add_kernel<<<blocks, threads, 0, stream>>>(input, output, size);

  // Check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA error in InplaceAdd: %s\n", hipGetErrorString(err));
  }
}

// Regular add for comparison (not in-place)
__global__ void regular_add_kernel(const float* a, const float* b,
                                   float* output, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    output[idx] = a[idx] + b[idx];
  }
}

void XlaGpuRegularAdd(hipStream_t stream, void** buffers, const char* opaque,
                      size_t opaque_len) {
  int32_t size;
  memcpy(&size, opaque, sizeof(int32_t));

  // buffers[0] = a, buffers[1] = b, buffers[2] = output
  const float* a = static_cast<const float*>(buffers[0]);
  const float* b = static_cast<const float*>(buffers[1]);
  float* output = static_cast<float*>(buffers[2]);

  printf("RegularAdd - A buffer:      %p\n", a);
  printf("RegularAdd - B buffer:      %p\n", b);
  printf("RegularAdd - Output buffer: %p\n", output);

  int threads = 256;
  int blocks = (size + threads - 1) / threads;
  regular_add_kernel<<<blocks, threads, 0, stream>>>(a, b, output, size);
}

}  // extern "C"